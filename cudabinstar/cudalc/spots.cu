
#include <hip/hip_runtime.h>
#include <math.h> 


__host__ __device__ void ld_quad_match(int ld_law, double * ldc, double * I_ret)
{
	// Set coefficients of a quadratic limb darkening law so that the intensity
	// profile matches at mu= 0, 0.5, 1.0
	// N.B.  these are the coefficients on the quadratic limb darkening law as used
	// in eker, i.e.,  I_0[1 - u_1.mu + u_2.mu^2], so u_2 is the negative of the
	// normal quadratic limb darkening coefficient.

	if (ld_law==-99)
	{
		I_ret[0] = 0.0;
		I_ret[1] = 0.0;
	}
	else if (ld_law==0)
	{
		I_ret[0] = ldc[0];
		I_ret[1] = 0.0;
	}
	else if (ld_law==1)
	{
		I_ret[0] = ldc[0];
		I_ret[1] = -ldc[1];		
	}
}




__host__ __device__ double eker_spots(double l, double b, double i, double r, double a, double u1, double u2, double phi)
{
    /*
    !
    ! Analytic light curve for a rotating star with a single circular spot 
    !
    ! Implementation of formulae from Eker, Z., 1994ApJ...420..373E with 
    ! correction for equation for K_3 from erratum 1994ApJ...430..438E.
    !
    ! Input values
    !  l        = longitude of spot centre (radians)
    !  b        = latitude of spot centre (radians)
    !  i        = inclination of rotation axis (radians)
    !  r        = angular radius of spot (radians)
    !  a        = Spot contrast ratio (a=Is/Ip).
    !  u1       = linear limb darkening coefficient.
    !  u2       = quadratic limb darkening coefficient.
    !  phi(nphi) = array of rotation phase values in radians, i.e. twopi*(t-t_0)/P
    !  nphi      = number of phase values
    !
    ! N.B. Eker uses a limb darkening law of the form I_0[1 - u_1.mu + u_2.mu^2], 
    ! i.e., u_2 is the negative of the normal quadratic limb darkening coefficient.
    !
    ! Output values
    !  df(nphi)  = light curve
    !  ii(nphi)  = spot position flag
    !  ifail    = status flag
    !
    ! Return values in the array df are (Fp + Fs)/F where
    !  Fp is the flux from the unspotted photosphere
    !  Fs is the flux from the spot
    !  F is the flux from the star without a spot.
    ! 
    ! Spot position flag ii() is as follows
    !   0 = spot not visible
    !   1 = Spot is on the limb and less than half the spot is visible
    !   2 = Spot is on the limb and more than half the spot is visible.
    !   3 = Spot is completely visible
    !
    ! Return value of ifail is the sum of the following flags.
    !   0 => all ok
    !   1 => r >= pi/2
    !   2 => a < 0.0
    !   4 => nphi < 1
    !
    */

    // Local variables 
    double cosisinb,sinicosb;
    double sinr2,sinr3,sinr4,sin2r;
    double cosr,cos2r,cosr3,cosr4;
    double qn,ql,qq;
    double th0,Cl,Cq,K1,K2,K3,K4;
    double t13b1,t13b2,t13c1,t13c2;
    double costh0,sinth0,sinth02,tanth0;
    double phi0,r0,sinphi0,cosphi0,sinr0,cosr0,sin2r0,cos2r0;
    double f0,fn,fl,fq;

    int ifail = 0;
    double pi = M_PI;
    double halfpi = M_PI/2.; 

    // First check
    if (r >= halfpi) ifail = ifail + 1;
    if (a < 0.0)  ifail = ifail + 2;
    if (ifail != 0) return 0. ;

    // constants 
    cosisinb = cos(i)*sin(b);
    sinicosb = sin(i)*cos(b);
    cosr = cos(r);
    cosr3 = pow(cosr,3);
    cosr4 = pow(cosr,4);
    sin2r = sin(2.0*r);
    cos2r = cos(2.0*r);
    sinr2 = 1.0 - pow(cosr,2);
    sinr3  = pow(sin(r),3);
    sinr4  = pow(sinr2,2);
    f0 = (a-1.0)/(1.0-u1/3.0+u2/6.0);
    fn = (1.0-u1+u2)*f0;
    fl = (u1-2.0*u2)*f0;
    fq = u2*f0;
    t13b1 = 2.0*(1.0-cosr3)/3.0;
    t13b2 = cosr*sinr2;
    t13c1 = 0.5*(1.0-cosr4);
    t13c2 = 0.75*sinr4;

    costh0 = cosisinb+sinicosb*cos(phi-l);
    th0 = acos(costh0);
    sinth02 = 1.0 - pow(costh0,2);
    sinth0 = sqrt(sinth02);
    tanth0 = sinth0/costh0;

    // First, check to make sure the spot isn't around the back 
    if (th0 >= (halfpi+r)) return 0.;
    else
    {
        // Here, the spot is partly or fully visible
        if (u1 == 0.0 & u2 == 0.0)
        {
            qn = sinr2*costh0;
            ql = 0.0;
            qq = 0.0;
        }
        else if (u2==0.)
        {
            qn = sinr2*costh0;
            ql = t13b1 - t13b2*sinth02;
            qq = 0.0;          
        }
        else
        {
            qn = sinr2*costh0;
            ql = t13b1 - t13b2*sinth02;
            qq = t13c1*pow(costh0,3) + t13c2*costh0*sinth02;
        }

        if (th0 > halfpi)
        {
            cosphi0 = -1.0/(tanth0*tan(r));
            phi0 = acos(cosphi0);
            sinphi0 = sin(phi0);
            qn = (phi0*qn - asin(cosr/sinth0) - 0.5*sinth0*sinphi0*sin2r)/pi + 0.5;
            
            if (u1 != 0.0 | u2 != 0.0)
            {
                r0 = abs(th0-halfpi);
                sinr0 = sin(r0);
                cosr0 = cos(r0);
                sin2r0 = sin(2.0*r0);
                cos2r0 = cos(2.0*r0);
                // (19a)
                ql = (phi0/3.0*(cosr3-pow(cosr0,3))* (1.0-3.0*pow(costh0,2)) - (phi0 + sinphi0*cosphi0)*(cosr - cosr0)*sinth02 + 4.0/3.0*sinphi0*(sinr3-pow(sinr0,3))*sinth0*costh0 + sinphi0*cosphi0/3.0*(cosr3 - pow(cosr0,3))*sinth02)/pi;
                if (u2 != 0.0) 
                {
                    K1 = 0.25*phi0*(pow(cosr0,4)-cosr4);
                    K2 = -0.125*sinphi0*(r0 - r + 0.5*(sin2r*cos2r-sin2r0*cos2r0));
                    K3 = 0.125*(phi0+sinphi0*cosphi0)*(sinr4-pow(sinr0,4));
                    K4 = (sinphi0-pow(sinphi0,3)/3.0)*(0.375*(r-r0) + 0.0625*(sin2r*(cos2r-4.0)-sin2r0*(cos2r0-4.0)));
                    qq = (2.0*pow(costh0,3)*K1 + 6.0*costh0*sinth0* (costh0*K2 + sinth0*K3) +  2.0*pow(sinth0,3)*K4 )/pi;     
                }  
            }
        } 
        else if (th0 > (halfpi-r)) // Spot is on the limb and more than half the spot is visible..
        {
            cosphi0 = -1.0/(tanth0*tan(r));
            phi0 = acos(cosphi0);
            sinphi0 = sin(phi0);
            qn = (phi0*qn - asin(cosr/sinth0) - 0.5*sinth0*sinphi0*sin2r)/pi + 0.5;

            if ((u1 != 0.0 | u2 != 0.0))
            {
                r0 = abs(th0-halfpi);
                sinr0 = sin(r0);
                cosr0 = cos(r0);
                sin2r0 = sin(2.0*r0);
                cos2r0 = cos(2.0*r0);
                // (18a)
                Cl = (pi-phi0)/3.0*(cosr3-pow(cosr0,3))*(1.0-3.0*pow(costh0,2)) - (pi-phi0-sinphi0*cosphi0)*(cosr-cosr0)*sinth02 - 4.0/3.0*sinphi0*(sinr3-pow(sinr0,3))*sinth0*costh0 - sinphi0*cosphi0*(cosr3-pow(cosr0,3))*sinth02/3.0;
                ql = ql - Cl/pi;
                if (u2 != 0.0)
                {
                    K1 = 0.25*(pi-phi0)*(pow(cosr0,4)-cosr4);
                    K2 = 0.125*sinphi0*(r0 - r + 0.5*(sin2r*cos2r-sin2r0*cos2r0));
                    K3 = 0.125*(pi-phi0+sinphi0*cosphi0)*(sinr4-pow(sinr0,4));
                    K4 = -(sinphi0-pow(sinphi0,3)/3.0)*(0.375*(r-r0) + 0.0625*(sin2r*(cos2r-4.0)-sin2r0*(cos2r0-4.0)));
                    // (18b)
                    Cq = 2.0*pow(costh0,3)*K1 + 6.0*costh0*sinth0*(costh0*K2 + sinth0*K3) + 2.0*pow(sinth0,3)*K4;
                    qq = qq - Cq/pi;
                }
            }
        }
        return fn*qn + fl*ql + fq*qq;
    }
}
