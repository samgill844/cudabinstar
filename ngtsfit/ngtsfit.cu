#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctype.h>
#include <unistd.h>
#include <getopt.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <locale.h>

#include "../include/sampler_helper.h"
#include "../include/ensemble.h"
#include "../include/lc.h"

/* Flag set by ‘--verbose’. */
static int verbose_flag;

#define MAXCHAR 1000


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


int count_3_col(char * filename, double t_zero, double period, double pcut)
{

    // count the number of lines in the file called filename                                    
    FILE *fp = fopen(filename,"r");
    if (fp == NULL)
    {
        if (!fp)perror("fopen");
        printf("\nI've failed :(");
    }  

    char str[MAXCHAR];

    double a,b,c,d,e, phase;
    int count=0;
    while (fgets(str, MAXCHAR, fp) != NULL)
    {
        sscanf(str, "%lf %lf %lf %lf %lf", &a, &b, &c, &d, &e);
        phase = ((a - t_zero)/period) - floor((a - t_zero)/period);
        if (phase < pcut || phase > (1-pcut)) count ++;
    }
        //printf("%s", str);
    fclose(fp);
    return count;
}

void readlread_3_col(char * filename, int lines_to_read, double * x, double * y, double * z, double t_zero, double period, double pcut)
{

    // count the number of lines in the file called filename                                    
    FILE *fp = fopen(filename,"r");
    if (fp == NULL)
    {
        if (!fp)perror("fopen");
        printf("\nI've failed :(");
    }  

    char str[MAXCHAR];

    double a,b,c,d,e, phase;
    int count=0;
    while (fgets(str, MAXCHAR, fp) != NULL)
    {
        sscanf(str, "%lf %lf %lf %lf %lf", &a, &b, &c, &d, &e);
        phase = ((a - t_zero)/period) - floor((a - t_zero)/period);
        if (phase < pcut || phase > (1-pcut))
        {
            x[count]=a;
            y[count]=b;
            z[count]=c;
            count ++;
        }
    }
        //printf("%s", str);
    fclose(fp);
}




int main(int argc, char* argv[])
{
    printf("\n------------------------------------");
    printf("\n-          NGTSfit V0.1            -");
    printf("\n-      samgill844@gmail.com        -");
    printf("\n------------------------------------");
    printf("\e[?25l"); // stop blinking cursor

    // Filename
    char *input_filename = "ngts.lc";
    char *output_filename = "output.dat";


    // Lightcurve parameters
    double t_zero = 0.0;
    double period = 1.0;
    double pcut = 0.1;
    double radius_1 = 0.2;
    double k = 0.2;
    double zp = 0.;
    double jitter = 0.001;
    double b = 0.1;

    // Limb-darkening parameters
    int ld_law = 0;
    double ldc_1 = 0.65;
    double ldc_2 = 0.37;

    // Fitting parameters
    int nsteps = 1000;
    int burn_in = 950;
    int nwalkers = 10240;
    int threads_per_block = 256;

    // GPU or CPU
    int CPU_OR_GPU = 0; // 0 = CPU, 1 = GPU
    int device = 0; // the GPU device

    // Need to re-jig this according to
    // https://www.gnu.org/software/libc/manual/html_node/Getopt-Long-Option-Example.html#Getopt-Long-Option-Example
    opterr = 0;

    while (1)
    {
        static struct option long_options[] =
        {
            /* These options set a flag. */
            {"verbose", no_argument,       &verbose_flag, 1},
            {"brief",   no_argument,       &verbose_flag, 0},
            {"gpu", no_argument,       &CPU_OR_GPU, 1},

            /* These options don’t set a flag.
                We distinguish them by their indices. */
            {"filename",     required_argument,       0, 'f'},
            {"output",       required_argument,       0, 'o'},
            {"t_zero",       required_argument,       0, 't'},
            {"period",       required_argument,       0, 'p'},
            {"pcut",         required_argument,       0, 'c'},
            {"radius_1",     required_argument,       0, 'r'},
            {"k",            required_argument,       0, 'k'},
            {"zp",            required_argument,       0, 'z'},
            {"jitter",            required_argument,  0, 'j'},
            {"impact",            required_argument,  0, 'u'},

            {"ld_1",         required_argument,       0, 'l'},
            {"ldc_1",        required_argument,       0, 'q'},
            {"ldc_2",        required_argument,       0, 'w'},

            {"nsteps",       required_argument,       0, 'n'},
            {"burn_in",      required_argument,       0, 'b'},
            {"walkers",      required_argument,       0, 'e'},
            {"threads_per_block", required_argument,       0, 'y'},

            {"device",       required_argument,       0, 'd'},
            {0, 0, 0, 0}
        };

        /* getopt_long stores the option index here. */
        int option_index = 0;

        int c = getopt_long (argc, argv, "f:o:t:p:c:r:k:l:q:w:n:b:w:y:d:",
                        long_options, &option_index);
    
        /* Detect the end of the options. */
        if (c == -1) break;   


        switch (c)
        {
        case 0:
          /* If this option set a flag, do nothing else now. */
          if (long_options[option_index].flag != 0) break;
          printf ("option %s", long_options[option_index].name);
          if (optarg) printf (" with arg %s", optarg);
          printf ("\n");
          break;

        case 'f':
            input_filename = optarg;
            break;

        case 'o':
            output_filename = optarg;
            break;

        case 't':
            sscanf(optarg,"%lf",&t_zero);
            break;

        case 'p':
            sscanf(optarg,"%lf",&period);
            break;
     
        case 'c':
            sscanf(optarg,"%lf",&pcut);
            break;
    
        case 'r':
            sscanf(optarg,"%lf",&radius_1);
            break;

        case 'k':
            sscanf(optarg,"%lf",&k);
            break;

        case 'z':
            sscanf(optarg,"%lf",&zp);
            break;

        case 'j':
            sscanf(optarg,"%lf",&jitter);
            break;

        case 'u':
            sscanf(optarg,"%lf",&b);
            break;

        case 'l':
            sscanf(optarg,"%d",&ld_law);
            break;

        case 'q':
            sscanf(optarg,"%lf",&ldc_1);
            break;

        case 'w':
            sscanf(optarg,"%lf",&ldc_2);
            break;

        case 'n':
            sscanf(optarg,"%d",&nsteps);
            break;

        case 'b':
            sscanf(optarg,"%d",&burn_in);
            break;

        case 'e':
            sscanf(optarg,"%d",&nwalkers);
            break;

        case 'y':
            sscanf(optarg,"%d",&threads_per_block);
            break;

        case 'd':
            sscanf(optarg,"%d",&device);
            break;

        case '?':
          /* getopt_long already printed an error message. */
          break;

        default:
          abort ();
        }

    }
    
    printf("\nExample use:");
    printf("\nngtsfit [filename] [t_zero] [period] [pcut] [radius_1=0.2] [k=0.2] [h1=0.65] [h2=0.37]");
    printf("\n\t\t[nsteps=1000] [burn_in=950] [nwalkers=10240] [threads_per_block=256]");
    printf("\n\t\t[output_file=NGTSfit_results.dat] [gpu // cpu]\n\n");

    int blocks = (int) ceil(nwalkers/threads_per_block);
    

    /*---------------------------
     Part 0 - report choices
     ---------------------------*/
    printf("\nFitting parameters:");
    printf("\n\tnsteps : %d", nsteps); fflush(stdout);
    printf("\n\tburn in : %d", burn_in); fflush(stdout);
    printf("\n\tnwalkers : %d", nwalkers); fflush(stdout);
    printf("\n\tpcut : %f", pcut); fflush(stdout);

    printf("\n\tthreads_per_block : %d", threads_per_block); fflush(stdout);
    printf("\n\tblocks: %d", blocks); fflush(stdout);
    printf("\n\tinput file : %s", input_filename);
    printf("\n\toutput file: %s", output_filename); fflush(stdout);
    printf("\n\tdevice used: "); fflush(stdout);
    switch(CPU_OR_GPU){ case 1 : printf("GPU\n"); break;  case 0 : printf("CPU\n"); break;}

    printf("\n\tt_zero : %f", t_zero);
    printf("\n\tperiod : %f", period);
    printf("\n\tradius_1 : %f", radius_1);
    printf("\n\tk : %f", k);
    printf("\n\tzp : %f", zp);
    printf("\n\tjitter : %f", jitter);
    printf("\n\timpact : %f", b);

    printf("\n\tldc_1 : %f", ldc_1);
    printf("\n\tldc_2 : %f", ldc_2);


    /*---------------------------
     Part 1 - read the LC 
     ---------------------------*/
    printf("\n\nReading data from %s:", input_filename);
    double *time, *d_time, *LC, *d_LC, *LC_ERR, *d_LC_ERR, *d_N_LC;
    int N_LC = count_3_col(input_filename, t_zero, period, pcut);
    printf("\n\tNumber of lines : %d", N_LC);fflush(stdout);
    time = (double *) malloc(N_LC*sizeof(double));
    LC = (double *) malloc(N_LC*sizeof(double));
    LC_ERR = (double *) malloc(N_LC*sizeof(double));
    readlread_3_col(input_filename, N_LC, time, LC, LC_ERR,  t_zero, period, pcut);
    printf("\n\tRead in OK!");fflush(stdout);


    /*---------------------------
     Part 2 - initialise theta
     ---------------------------*/
    double * theta;
    int ndim=9;
    theta = (double *) malloc(ndim*sizeof(double));
    // 0 : t_zero
    // 1 : period
    // 2 : radius_1
    // 3 : k
    // 4 : h1
    // 5 : h2
    // 6 : b
    // 7 : zp
    // 8 : J

    theta[0] = t_zero;
    theta[1] = period;
    theta[2] = radius_1;
    theta[3] =  k;
    theta[4] = ldc_1;
    theta[5] =  ldc_2;
    theta[6] = b;
    theta[7] =  zp;
    theta[8] =  jitter;

    double loglik =     lc_loglike(time, LC, LC_ERR, theta[7],theta[8],
        theta[0], theta[1],
        theta[2], theta[3] ,
        0., 0., 
        90.,
        0, theta[4], theta[5], 
        0., 0.,
        0, 0.001, 0, 0.001,
        N_LC );
    printf("\n\n------------------------------\nInitial loglike : %f\n------------------------------\n", loglik);

    /*---------------------------
    Part 3 - configure the arguments for either GPU ot CPU
    ---------------------------*/
    printf("Configuring arguments for the "); fflush(stdout);
    double ** args, **d_args;
    switch(CPU_OR_GPU){ case 1 : printf("GPU... "); break;  case 0 : printf("CPU... "); break;}

    double * tmpp;
    tmpp = (double *) malloc(5*sizeof(double));
    tmpp[0] = (double) N_LC;
    tmpp[1] = t_zero;
    tmpp[2] = period;
    tmpp[3] = ldc_1;
    tmpp[4] = ldc_2;
    switch(CPU_OR_GPU){ 
                        
        case 0:
            args = (double **) malloc(4*sizeof(double *));
            args[0] = time;
            args[1] = LC;
            args[2] = LC_ERR;
            args[3] = tmpp;
            break;
        
        case 1 :
                    // Now set up the args
                    hipMalloc(&d_time, N_LC*sizeof(double)); 
                    hipMalloc(&d_LC, N_LC*sizeof(double)); 
                    hipMalloc(&d_LC_ERR, N_LC*sizeof(double)); 
                    hipMalloc(&d_N_LC, 5*sizeof(double)); 

                    hipMemcpy(d_time, time, N_LC*sizeof(double), hipMemcpyHostToDevice);
                    hipMemcpy(d_LC, LC, N_LC*sizeof(double), hipMemcpyHostToDevice);
                    hipMemcpy(d_LC_ERR, LC_ERR, N_LC*sizeof(double), hipMemcpyHostToDevice);
                    hipMemcpy(d_N_LC, tmpp, 5*sizeof(double), hipMemcpyHostToDevice);

                    args = (double **) malloc(4*sizeof(double *));
                    args[0] = d_time;
                    args[1] = d_LC;
                    args[2] = d_LC_ERR;
                    args[3] = d_N_LC;
                    hipMalloc(&d_args, 4*sizeof(double**)); 
                    hipMemcpy(d_args, args, 4*sizeof(double**), hipMemcpyHostToDevice);
                    break;

                }
    printf("done."); fflush(stdout);



    /*---------------------------
    Part 4 - Create the starting positions
    ---------------------------*/
    printf("\nCreating the sarting positions."); fflush(stdout);
    double scatter = 0.0001 ;
    double * d_positions, * d_loglikliehoods;
    double * positions, *loglikliehoods ;
    switch(CPU_OR_GPU)
    {
        case 0:
            int j,k;
            loglikliehoods = (double *) malloc(nwalkers*nsteps*sizeof(double));
            positions = (double*) malloc(nwalkers*nsteps*ndim*sizeof(double));
            for (j=0; j < nwalkers; j++)
            {
                for (k=0;  k < ndim; k++)
                {
                    positions[j*ndim + k] = theta[k] + scatter*sampleNormal_d();
                }
            }
            printf(" done."); fflush(stdout);
            break    ;  

        case 1:
            // First malloc the device arrays
            gpuErrchk(hipMalloc(&d_positions, nwalkers*nsteps*ndim*sizeof(double))); 
            gpuErrchk(hipMalloc(&d_loglikliehoods, nwalkers*nsteps*sizeof(double))); 

            create_starting_positions(theta,
                nsteps, ndim, nwalkers,
                blocks, threads_per_block,
                scatter, 
                d_positions,
                1);
            printf(" done."); fflush(stdout);
            break;   
    }

    /*---------------------------
    Part 5 - Create the hiprand states if requireds
    ---------------------------*/
    hiprandState *devState;
    if (CPU_OR_GPU==1)
    {
        // Create the hiprand states
        printf("\nCreating the hiprand states... "); fflush(stdout);
        gpuErrchk(hipMalloc((void**)&devState, nwalkers*sizeof(hiprandState)));
        initCurand<<<ceil(nwalkers/256),256>>>(devState, 1);
        printf(" done."); fflush(stdout);
    }

    /*---------------------------
    Part 5 - Create the blocks to monitor progress
    ---------------------------*/
    int * d_block_progress;
    int i;
    
    if (CPU_OR_GPU==1)
    {
        int d_block_progress__[1] = {0};
        gpuErrchk(hipMalloc(&d_block_progress, blocks*sizeof(int))); 
        for (i=0; i < blocks; i++) gpuErrchk(hipMemcpy(&d_block_progress[i], &d_block_progress__, sizeof(int), hipMemcpyHostToDevice));
    }


    /*---------------------------
    Part 5 - Launch the sampler
    ---------------------------*/
    clock_t start, diff;
    switch (CPU_OR_GPU)
    {
        case 0 :
            // Now run
            printf("\n-----------------------------------");
            printf("\nCommencing Bayesian sampleing [CPU]\n"); fflush(stdout);
            CPU_parallel_stretch_move_sampler(nsteps, ndim, nwalkers, args, 
                loglikliehoods, positions,
                2.0);
            break;

        case 1 :
            hipStream_t streams[2];
            hipStreamCreate(&streams[0]);
            hipStreamCreate(&streams[1]);


            // Now run
            printf("\n-----------------------------------");
            printf("\nCommencing Bayesian sampleing [GPU]\n"); fflush(stdout);
            printf("Progress of each block [%%] given below..."); 

            // Start the progress bar
            sampler_progress<<<1, 1, 0,streams[0] >>>(blocks, d_block_progress);

            start = clock();
            GPU_parallel_stretch_move_sampler<<<blocks, threads_per_block, 0 , streams[1]>>>(nsteps, ndim, nwalkers, blocks, threads_per_block, d_args, 
                d_loglikliehoods, d_positions,
                2.0,  devState, d_block_progress );
            hipGetLastError();
            hipDeviceSynchronize(); // make sure the kernel is done before carrying on.

            printf("\n-----------------------------------");fflush(stdout);
            diff = clock() - start;
            int msec = diff * 1000 / CLOCKS_PER_SEC;
            int number_of_models_per_second = nsteps*nwalkers/ (msec/1000);
            setlocale(LC_NUMERIC, "");
            printf("\nTime taken %'d seconds %'d milliseconds", msec/1000, msec%1000);
            printf("\nNumber of models per second : %'d", number_of_models_per_second);
            printf("\nNumber of models per minute : %'d", 60*number_of_models_per_second);
            printf("\n-----------------------------------");fflush(stdout);
            break;
    }
    

    /*---------------------------
    Part 6 - Write out results
    ---------------------------*/
    // Write out
    printf("\nWriting results... "); fflush(stdout);
    switch (CPU_OR_GPU)
    {
        case 0 :
            write_out_results(burn_in, nsteps, ndim, nwalkers,
                blocks, threads_per_block,
                positions, loglikliehoods, output_filename, CPU_OR_GPU);
            break;
        case 1 :
            write_out_results(burn_in, nsteps, ndim, nwalkers,
                blocks, threads_per_block,
                d_positions, d_loglikliehoods, output_filename, CPU_OR_GPU);
            break;
    }

    printf(" done.\n\n"); fflush(stdout);
    
    
    
    // free up host memory
    free(theta);
    free(args);
    free(time);
    free(LC);
    free(LC_ERR);
    hipFree(d_time);
    hipFree(d_LC);
    hipFree(d_LC_ERR);


    // free up device memory
    if (CPU_OR_GPU)
    {
        hipFree(d_positions);
        hipFree(d_loglikliehoods);
        hipFree(devState);
        hipFree(d_time);
        hipFree(d_LC);
        hipFree(d_LC_ERR);
        hipFree(d_args);
        hipFree(d_block_progress);
    }

    printf("\e[?25h"); // show the cursor again
    
    //historgram_plot_2_axis();
    
    
}